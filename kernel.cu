#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "utilities.cuh"
#include <stdio.h>
#include <stdlib.h>

__global__ void addKernel(int *c, int *a, int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
 /*   int a[5] = { 1, 2, 3, 4, 5 };
    int b[5] = { 10, 20, 30, 40, 50 };
    int c[5] = { 0 };*/

    int* a, * b, * c;
    size_t size = 5 * sizeof(int);
    a = (int*)malloc(size);
    b = (int*)malloc(size);
    c = (int*)malloc(size);

    a[0] = 1; a[1] = 2; a[2] = 3; a[3] = 4; a[4] = 5;
    b[0] = 10; b[1] = 20; b[2] = 30; b[3] = 40; b[4] = 50;
    c[0] = 0; c[1] = 0; c[2] = 0; c[3] = 0; c[4] = 0; 

    hipError_t cudaStatus;

    int* a_d = NULL;
    int* b_d = NULL;
    int* c_d = NULL;

    int** array_h[3] = { &a, &b, &c };
    int** array_d[3] = { &a_d, &b_d, &c_d };
    int** array[3] = { &a_d, &b_d, &c_d };


    cudaStatus = arrayMalloc((void***)array, 3, size);

    /*hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, size, hipMemcpyHostToDevice);
    hipMemcpy(c_d, c, size, hipMemcpyHostToDevice);*/

    /*for (int i = 0; i < 3; i++)
    {
        hipMemcpy(*array[i], array_h[i], size, hipMemcpyHostToDevice);
        if (array_d[i] == NULL)
        {
            fprintf(stderr, "array[%d] is NULL\n", i);
        }
    }*/

    cudaStatus = arraycpyHtoD((void***)array_d, (void***)array_h, 3, size);

    addKernel <<<1, 5 >>> (c_d, a_d, b_d);

    cudaStatus = onecpyDtoH((void*)c, (void*)c_d, size);


    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
 /*   cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }*/

    return 0;
}

//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;

//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
