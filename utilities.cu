#include "utilities.cuh"


hipError_t arrayMalloc(void*** array, int length, size_t size)
{
	hipError_t cudaStatus;

	for (int i = 0; i < length; i++)
	{
		cudaStatus = hipMalloc(array[i], size);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMalloc failed!\n");
			goto Error;
		}
	}
Error:
	for (size_t i = 0; i < length; i++)
	{
		hipFree(array[i]);
	}

	return cudaStatus;
}

hipError_t arraycpyHtoD(void*** array_d, void*** array_h, int length, size_t size)
{
	hipError_t cudaStatus;

	for (int i = 0; i < length; i++)
	{
		cudaStatus = hipMemcpy(*array_d[i], *array_h[i], size, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy HostToDevice of array %d failed!\n", i);
			fprintf(stderr, "Reasons for failure : %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}
	}
Error:
	for (size_t i = 0; i < length; i++)
	{
		hipFree(array_d[i]);
	}

	return cudaStatus;
}

hipError_t arraycpyDtoH(void*** array_h, void*** array_d, int length, size_t size)
{
	hipError_t cudaStatus;

	for (int i = 0; i < length; i++)
	{
		cudaStatus = hipMemcpy(*array_h[i], *array_d[i], size, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess)
		{
			fprintf(stderr, "hipMemcpy DeviceToHost of array %d failed!\n", i);
			goto Error;
		}
	}
Error:
	for (size_t i = 0; i < length; i++)
	{
		hipFree(array_d[i]);
	}

	return cudaStatus;
}

hipError_t oneMalloc(void** a_d, size_t size)
{
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&a_d, size );
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
	}
	return cudaStatus;
}

hipError_t onecpyHtoD(void* dev_a, void* a, size_t size)
{
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
	}
	return cudaStatus;
}

hipError_t onecpyDtoH(void* a, void* dev_a, size_t size)
{
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(a, dev_a, size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
	}
	return cudaStatus;
}

hipError_t oneSetdevice()
{
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
	}
	return cudaStatus;
}

hipError_t oneLastError()
{
	hipError_t cudaStatus;
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
	return cudaStatus;
}

// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.

hipError_t oneCudaDeviceSync()
{
	hipError_t cudaStatus;
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}
	return cudaStatus;
}